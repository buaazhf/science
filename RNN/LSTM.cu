#include "hip/hip_runtime.h"
#include "mex.h"
#include "stdio.h"
#include <string.h>
#define blocksize 32
#define THREAD_NUM 512
#define BLOCK_NUM 256

#define eps 1.0e-8

__global__ void Mul(float *W,float *X,float *output,int L_rows,int L_cols,int R_cols)
{

    int bid=blockIdx.x;
    int row=threadIdx.y;
    int col=threadIdx.x;
    int blockRow,blockCol,r=(L_rows+blocksize-1)/blocksize,c=(R_cols+blocksize-1)/blocksize;
    float sum;

for(int u=bid;u<r*c;u+= BLOCK_NUM)
{  
   sum=0;
   blockRow=u%r;
   blockCol=u/r;
   
for(int i=0;i<((L_cols+blocksize-1)/blocksize);i++)
{

__shared__ float subA[blocksize][blocksize];
__shared__ float subB[blocksize][blocksize];

if((blockRow*blocksize+row)<L_rows&&(i*blocksize+col)<L_cols)
subA[row][col]=W[(i*blocksize+col)*L_rows+blockRow*blocksize+row];
else
subA[row][col]=0;
if((blockCol*blocksize+col)<R_cols&&(i*blocksize+row)<L_cols)
subB[row][col]=X[L_cols*(blockCol*blocksize+col)+row+i*blocksize];
else
subB[row][col]=0;

__syncthreads(); 
for(int j=0;j<blocksize;j++)
   sum+=subA[row][j]*subB[j][col];
__syncthreads(); 
} 
if((blockRow*blocksize+row)<L_rows&&(blockCol*blocksize+col)<R_cols)

output[L_rows*(blockCol*blocksize+col)+blockRow*blocksize+row]=sum;


}
}

__global__  void Active(float *output_x,float *output_a1,float *b1,float *ft1,float *it1,float *cct1,float *ot1,int sum)

{
   const int tid=threadIdx.x;
   const int bid=blockIdx.x;
   int n_a=128,p,q,n_a4=4*128;
   for(int u=tid+bid*THREAD_NUM;u<sum;u+=BLOCK_NUM*THREAD_NUM)
    { p=u/n_a4;
      q=u%n_a4;
      if(q<n_a)
        ft1[q+p*n_a]=1/(1+exp(-(output_x[u]+output_a1[u]+b1[q])));
      else if(q>=n_a&&q<(2*n_a))
        it1[q-n_a+p*n_a]=1/(1+exp(-(output_x[u]+output_a1[u]+b1[q])));
      else if(q>=(2*n_a)&&q<(3*n_a))
        cct1[q-2*n_a+p*n_a]=2/(1+exp(-2*(output_x[u]+output_a1[u]+b1[q])))-1;
      else
        ot1[q-3*n_a+p*n_a]=1/(1+exp(-(output_x[u]+output_a1[u]+b1[q])));
     }

}
__global__ void pointwise(float *ft1,float *it1,float *cct1,float *ot1,float *a_next1,float *c_next1,float *c_prev1,int sum)

{
   const int tid=threadIdx.x;
   const int bid=blockIdx.x;

   for(int u=tid+bid*THREAD_NUM;u<sum;u+=BLOCK_NUM*THREAD_NUM)
{
 c_next1[u]=ft1[u]*c_prev1[u]+it1[u]*cct1[u];
 a_next1[u]=ot1[u]*(2/(1+exp(-2*c_next1[u]))-1);
}

}

__global__ void softmax(float *W,float *X,float *output,int L_rows,int L_cols,int R_cols,float *by)
{

    int bid=blockIdx.x;
    int row=threadIdx.y;
    int col=threadIdx.x;
    int blockRow,blockCol,r=(L_rows+blocksize-1)/blocksize,c=(R_cols+blocksize-1)/blocksize;
    float sum;

for(int u=bid;u<r*c;u+= BLOCK_NUM)
{  
   sum=0;
   blockRow=u%r;
   blockCol=u/r;
   
for(int i=0;i<((L_cols+blocksize-1)/blocksize);i++)
{

__shared__ float subA[blocksize][blocksize];
__shared__ float subB[blocksize][blocksize];

if((blockRow*blocksize+row)<L_rows&&(i*blocksize+col)<L_cols)
subA[row][col]=W[(i*blocksize+col)*L_rows+blockRow*blocksize+row];
else
subA[row][col]=0;
if((blockCol*blocksize+col)<R_cols&&(i*blocksize+row)<L_cols)
subB[row][col]=X[L_cols*(blockCol*blocksize+col)+row+i*blocksize];
else
subB[row][col]=0;

__syncthreads(); 
for(int j=0;j<blocksize;j++)
   sum+=subA[row][j]*subB[j][col];
__syncthreads(); 
} 
if((blockRow*blocksize+row)<L_rows&&(blockCol*blocksize+col)<R_cols)

output[L_rows*(blockCol*blocksize+col)+blockRow*blocksize+row]=exp(sum+by[blockRow*blocksize+row]);


}
}
__global__ void add(float *a,float *b,int n_y,int m)
{


   const int tid=threadIdx.x;
   const int bid=blockIdx.x;
   int offset=1,mask=1;
   __shared__ float shared[THREAD_NUM];
   shared[tid]=0;  
   for(int u=tid+bid*n_y;u<n_y*(bid+1);u+= 1)
    {
     shared[tid]+=a[u];

    }
    while(offset<THREAD_NUM)
   {
		if (tid&mask == 0) {
			shared[tid] += shared[tid + offset];
		}
		offset += offset;
        mask=offset+mask;
		__syncthreads();
	}
    if(tid==0)
    {
     b[bid]=shared[0];
     }


}
__global__ void  out(float *a,float *b,float *y_pred,float *output_diff,float *y_t,int sum)

{
   const int tid=threadIdx.x;
   const int bid=blockIdx.x;
   int r;
   for(int u=tid+bid*THREAD_NUM;u<sum;u+=BLOCK_NUM*THREAD_NUM)
{  r=u/6110;
   y_pred[u]=a[u]/b[r];
   if((u%6110)==(y_t[r]-1))
   output_diff[u]=1-y_pred[u];
   else
   output_diff[u]=-y_pred[u];

}

}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])

{  //[gradients,Allerror]=LSTM(train_x{num,1},train_y{num,1},parameters);

    
    const size_t *dim_array = mxGetDimensions(prhs[0]);
	int n_x=*dim_array,m=*(dim_array+1),T_x=*(dim_array+2);
    int n_a1=128,n_a2=128,n_y=6110;


    size_t  size_x=n_x*m*T_x*sizeof(float);
    size_t  size_y=m*T_x*sizeof(float);
    size_t  layer_1=n_a1*m*sizeof(float);
    size_t  layer_2=n_a2*m*sizeof(float);


    float *x_batch=(float*)mxGetPr(prhs[0]),*y_batch=(float*)mxGetPr(prhs[1]);

    float *a1,*c1,*a2,*c2,*x_t,*y_t;
    hipMalloc((void**)&a1,layer_1*(T_x+1));  
    hipMalloc((void**)&c1,layer_1*(T_x+1));
    hipMalloc((void**)&a2,layer_2*(T_x+1));
    hipMalloc((void**)&c2,layer_2*(T_x+1));

    hipMalloc((void**)&x_t,size_x);
    hipMalloc((void**)&y_t,size_y);

    hipMemset(a1,0,layer_1*(T_x+1));
    hipMemset(c1,0,layer_1*(T_x+1));
    hipMemset(a2,0,layer_2*(T_x+1));
    hipMemset(c2,0,layer_2*(T_x+1));
  
    hipMemcpy(x_t,x_batch,size_x,hipMemcpyHostToDevice);
    hipMemcpy(y_t,y_batch,size_y,hipMemcpyHostToDevice);


    float *host_w1_x=(float*)mxGetPr(mxGetField(prhs[2],0,mxGetFieldNameByNumber(prhs[2],0)));
    float *host_w1_a1=(float*)mxGetPr(mxGetField(prhs[2],0,mxGetFieldNameByNumber(prhs[2],1)));
    float *host_b1=(float*)mxGetPr(mxGetField(prhs[2],0,mxGetFieldNameByNumber(prhs[2],2)));
    float *host_w2_a1=(float*)mxGetPr(mxGetField(prhs[2],0,mxGetFieldNameByNumber(prhs[2],3)));
    float *host_w2_a2=(float*)mxGetPr(mxGetField(prhs[2],0,mxGetFieldNameByNumber(prhs[2],4)));
    float *host_b2=(float*)mxGetPr(mxGetField(prhs[2],0,mxGetFieldNameByNumber(prhs[2],5)));
    float *host_wy=(float*)mxGetPr(mxGetField(prhs[2],0,mxGetFieldNameByNumber(prhs[2],6)));
    float *host_by=(float*)mxGetPr(mxGetField(prhs[2],0,mxGetFieldNameByNumber(prhs[2],7)));

    float *w1_x,*w1_a1,*b1,*w2_a1,*w2_a2,*b2,*wy,*by;

    hipMalloc((void**)&w1_x,4*n_x*n_a1*sizeof(float));
    hipMalloc((void**)&w1_a1,4*n_a1*n_a1*sizeof(float));
    hipMalloc((void**)&b1,4*n_a1*sizeof(float));
    hipMalloc((void**)&w2_a1,4*n_a1*n_a2*sizeof(float));
    hipMalloc((void**)&w2_a2,4*n_a2*n_a2*sizeof(float));
    hipMalloc((void**)&b2,4*n_a2*sizeof(float));
    hipMalloc((void**)&wy,n_y*n_a2*sizeof(float));
    hipMalloc((void**)&by,n_y*sizeof(float));

    hipMemcpy(w1_x,host_w1_x,4*n_x*n_a1*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(w1_a1,host_w1_a1,4*n_a1*n_a1*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(b1,host_b1,4*n_a1*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(w2_a1,host_w2_a1,4*n_a1*n_a2*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(w2_a2,host_w2_a2,4*n_a2*n_a2*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(b2,host_b2,4*n_a2*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(wy,host_wy,n_y*n_a2*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(by,host_by,n_y*sizeof(float),hipMemcpyHostToDevice);



    float *output_1,*output_2,*output_3,*output_4,*output_5,*output_6,*ft1,*it1,*cct1,*ot1,*ft2,*it2,*cct2,*ot2;

    hipMalloc((void**)&output_1,4*n_a1*m*sizeof(float));  
    hipMalloc((void**)&output_2,4*n_a1*m*sizeof(float)); 
    hipMalloc((void**)&output_3,4*n_a2*m*sizeof(float));  
    hipMalloc((void**)&output_4,4*n_a2*m*sizeof(float)); 
    hipMalloc((void**)&output_5,n_y*m*sizeof(float)); 
    hipMalloc((void**)&output_6,m*sizeof(float)); 
    hipMemset(output_6,0,m*sizeof(float));
    hipMalloc((void**)&ft1,n_a1*m*sizeof(float)*T_x); 
    hipMalloc((void**)&it1,n_a1*m*sizeof(float)*T_x); 
    hipMalloc((void**)&cct1,n_a1*m*sizeof(float)*T_x); 
    hipMalloc((void**)&ot1,n_a1*m*sizeof(float)*T_x); 
    hipMalloc((void**)&ft2,n_a2*m*sizeof(float)*T_x); 
    hipMalloc((void**)&it2,n_a2*m*sizeof(float)*T_x); 
    hipMalloc((void**)&cct2,n_a2*m*sizeof(float)*T_x); 
    hipMalloc((void**)&ot2,n_a2*m*sizeof(float)*T_x); 
 
    float *y_pred,*output_diff,*da;

    hipMalloc((void**)&y_pred,n_y*m*sizeof(float)*T_x); 
    hipMalloc((void**)&output_diff,n_y*m*sizeof(float)*T_x); 
    
    hipStream_t streamA,streamB;
    hipStreamCreate(&streamA);
    hipStreamCreate(&streamB);

    for(int t=1;t<=T_x;t++){
       
        
       dim3 dimBlock(blocksize, blocksize);

        Mul<< <BLOCK_NUM,dimBlock,1024*4,streamA>> >(w1_x,x_t+(t-1)*n_x*m,output_1,4*n_a1,n_x,m);
        Mul<< <BLOCK_NUM,dimBlock,1024*4,streamB>> >(w1_a1,a1+(t-1)*n_a1*m,output_2,4*n_a1,n_a1,m);
      
        hipStreamSynchronize(streamA);
        hipStreamSynchronize(streamB);

        Active<< <BLOCK_NUM,THREAD_NUM>> >(output_1,output_2,b1,ft1+(t-1)*n_a1*m,it1+(t-1)*n_a1*m,cct1+(t-1)*n_a1*m,ot1+(t-1)*n_a1*m,4*n_a1*m);
        pointwise<< <BLOCK_NUM,THREAD_NUM>> >(ft1+(t-1)*n_a1*m,it1+(t-1)*n_a1*m,cct1+(t-1)*n_a1*m,ot1+(t-1)*n_a1*m,a1+t*n_a1*m,c1+t*n_a1*m,c1+(t-1)*n_a1*m,n_a1*m);
        
        Mul<< <BLOCK_NUM,dimBlock>> >(w2_a1,a1+t*n_a1*m,output_3,4*n_a2,n_a1,m);
        Mul<< <BLOCK_NUM,dimBlock>> >(w2_a2,a2+(t-1)*n_a2*m,output_4,4*n_a2,n_a2,m);

        Active<< <BLOCK_NUM,THREAD_NUM>> >(output_3,output_4,b2,ft2+(t-1)*n_a2*m,it2+(t-1)*n_a2*m,cct2+(t-1)*n_a2*m,ot2+(t-1)*n_a2*m,4*n_a2*m);
        pointwise<< <BLOCK_NUM,THREAD_NUM>> >(ft2+(t-1)*n_a2*m,it2+(t-1)*n_a2*m,cct2+(t-1)*n_a2*m,ot2+(t-1)*n_a2*m,a2+t*n_a2*m,c2+t*n_a2*m,c2+(t-1)*n_a2*m,n_a2*m);


        softmax<< <BLOCK_NUM,dimBlock>> >(wy,a2+t*n_a2*m,output_5,n_y,n_a2,m,by);
        add<< <m,THREAD_NUM>> >(output_5,output_6,n_y,m);
        out<< <BLOCK_NUM,THREAD_NUM>> >(output_5,output_6,y_pred+(t-1)*n_y*m,output_diff+(t-1)*n_y*m,y_t+(t-1)*m,n_y*m);
      }

    
     const size_t dim[]={n_y,m};
     plhs[0] = mxCreateNumericArray(2,dim ,mxSINGLE_CLASS, mxREAL);
     hipMemcpy((float*)mxGetPr(plhs[0]), y_pred+(T_x-1)*n_y*m, n_y*m*sizeof(float), hipMemcpyDeviceToHost);
     /*
     const size_t dim1[]={n_y,m};
     plhs[1] = mxCreateNumericArray(2,dim1 ,mxSINGLE_CLASS, mxREAL);
     hipMemcpy((float*)mxGetPr(plhs[1]), output_diff+(T_x-1)*n_y*m, n_y*m*sizeof(float), hipMemcpyDeviceToHost);
     */
     
     hipFree(a1);
     hipFree(c1);
     hipFree(a2);
     hipFree(c2);

     hipFree(x_t);
     hipFree(y_t);

     hipFree(y_pred);
     hipFree(output_diff);

     hipFree(w1_x);
     hipFree(w1_a1);
     hipFree(b1);
     hipFree(w2_a1);
     hipFree(w2_a2);
     hipFree(b2);
     hipFree(wy);
     hipFree(by);
     hipFree(output_1);
     hipFree(output_2);
     hipFree(output_3);
     hipFree(output_4);
     hipFree(output_5);
     hipFree(output_6);

     hipFree(ft1);
     hipFree(it1);
     hipFree(cct1);
     hipFree(ot1);
     hipFree(ft2);
     hipFree(it2);
     hipFree(cct2);
     hipFree(ot2);


	hipStreamDestroy(streamA);
	hipStreamDestroy(streamB);




    /*
    int nfields = mxGetNumberOfFields(prhs[2]);//��ȡ�ṹ���б����ĸ���
    printf("%d\n",nfields);
    //NStructElems = mxGetNumberOfElements(prhs[2]);//��ȡ�ṹ�������еĽṹ��ĸ���

    for (int ifield=0; ifield< nfields; ifield++){

       printf("%s\n",mxGetFieldNameByNumber(prhs[2],ifield));//��ȡ�����ṹ���ֶε�����

      }

     printf("%d\n",mxGetN(mxGetField(prhs[2],0,mxGetFieldNameByNumber(prhs[2],1))));//mxArray *mxGetField(const mxArray *pm, mwIndex index, const char *fieldname)
     */
    //���
    /*
    const size_t dim[]={y1,y2};
    plhs[0] = mxCreateNumericArray(number_of_dims1,dim ,mxSINGLE_CLASS, mxREAL);
    memcpy((float*)mxGetPr(plhs[0]), B, size_y);
    */
    /*
    double Allerror=7;
    plhs[1] = mxCreateDoubleMatrix(1,1,mxREAL); 
    *mxGetPr(plhs[1])=Allerror;

    mxArray  *fout;
    int a,b;
    const char *fieldnames[] = {"dw1","db1","dw2","db2","dwy","dby"};
    plhs[0]=mxCreateStructMatrix(1,1, nfields, fieldnames);

    for(int i=0;i<nfields;i++){
    a=mxGetM(mxGetField(prhs[2],0,mxGetFieldNameByNumber(prhs[2],i)));
    b=mxGetN(mxGetField(prhs[2],0,mxGetFieldNameByNumber(prhs[2],i)));
    
    const size_t dims[]={a,b};
    fout = mxCreateNumericArray(2, dims, mxSINGLE_CLASS, mxREAL);
    memcpy((float*)mxGetPr(fout), (float*)mxGetPr(mxGetField(prhs[2],0,mxGetFieldNameByNumber(prhs[2],i))),sizeof(float)*a*b);
    mxSetFieldByNumber(plhs[0], 0, i, fout);
    }
    */
}






