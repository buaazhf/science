#include "hip/hip_runtime.h"
#include "mex.h"
#include "stdio.h"
#include <string.h>
#include <time.h>
#include "hipblas.h"


#pragma comment(lib,"cublas.lib")

#define blocksize 32
#define THREAD_NUM 256
#define BLOCK_NUM 2048


__global__ void Im2col(float *In,float *Res_In,int a,int b,int c,int d,int height,int width,int batchsize,\
int In_channel,int output_channel,int pad_needed_height,int pad_needed_width,int new_height,int new_width)
{
   const int tid=threadIdx.x;
   const int bid=blockIdx.x;
   int i,k;
   int ii,jj,pp,qq,t;
   int index,flag;
   int padheight=pad_needed_height+height;
   for(int u=tid+bid*THREAD_NUM;u<c*d*In_channel*new_height*new_width*batchsize;u+= BLOCK_NUM*THREAD_NUM)
    {
        i=u/(new_height*new_width*batchsize);//λ������
        k=u%(new_height*new_width*batchsize);//λ������
        ii=k/(new_height*new_width);//λ���ĸ�batch
        jj=i/(c*d);  //λ���ĸ�In_channel
        pp=k%(new_height*new_width);
        qq=i%(c*d);
        index=(pp/new_height)*b*(height+pad_needed_height)+(pp%new_height)*a+(qq/c)*(height+pad_needed_height)+qq%c;
        if(index%padheight-pad_needed_height/2<0||index%padheight-pad_needed_height/2>=height||index/padheight-pad_needed_width/2<0||index/padheight-pad_needed_width/2>=width)
        Res_In[u]=0;
        else{
        flag=index%padheight-pad_needed_height/2+height*(index/padheight-pad_needed_width/2);
        t=jj*height*width*batchsize+ii*height*width+flag;
        Res_In[u]=In[t];
 
        }

     }

}

//�����
__global__ void Sigmod(float *input,float *output,int height,int width,int batchsize,int in_channel,float *bias)
{
   const int tid=threadIdx.x;
   const int bid=blockIdx.x;
   int r;
   for(int u=tid+bid*THREAD_NUM;u<height*width*batchsize*in_channel;u+=BLOCK_NUM*THREAD_NUM)
    {r=u/(height*width*batchsize);
     output[u]=1/(1+exp(-(input[u]+bias[r])));


    }

   
}
__global__ void Relu(float *input,float *output,int height,int width,int batchsize,int in_channel,float *bias)
{
   const int tid=threadIdx.x;
   const int bid=blockIdx.x;
   int r;
   for(int u=tid+bid*THREAD_NUM;u<height*width*batchsize*in_channel;u+=BLOCK_NUM*THREAD_NUM)
    { r=u/(height*width*batchsize);
       output[u]=max(0.0,input[u]+bias[r]);
      
    }

   
}
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])

{   /*output=conv2d(input,w,strides,padding,bias,Activfun)
      
      input=[height ,width ,batchsize ,in_channels]
      w=[filter_height , filter_width ,in_channels, output_channels]
      output=[height ,width ,batchsize ,output_channels]
      
      a=strides(1);b=strides(2);c=size(w,1);d=size(w,2);*/
    
    const size_t *dim_array = mxGetDimensions(prhs[0]);
	int height=*dim_array,width=*(dim_array+1),batchsize=1,In_channel=1;
    int number_of_dims = mxGetNumberOfDimensions(prhs[0]);
    if(number_of_dims==3)
     batchsize=*(dim_array+2);
    if(number_of_dims==4)
     {batchsize=*(dim_array+2);
      In_channel=*(dim_array+3);}

    const size_t *dim_array1 = mxGetDimensions(prhs[1]);
	int c=*dim_array1,d=*(dim_array1+1),output_channel=1;
    int number_of_dims1 = mxGetNumberOfDimensions(prhs[1]);
    if(number_of_dims1==4)
      output_channel=*(dim_array1+3);

    double *s;
    s=mxGetPr(prhs[2]);
    int a=int(*s),b=int(*(s+1));

    char *padding=mxArrayToString(prhs[3]);

    float *A=(float*)mxGetPr(prhs[0]);
    float *B=(float*)mxGetPr(prhs[1]);
    float *C=(float*)mxGetPr(prhs[4]);

    int new_height,new_width,pad_needed_height,pad_needed_width;

    if(strcmp(padding,"SAME")==0)
    {
     new_height= (height+a-1)/a;
     new_width=(width+b-1)/b;
     pad_needed_height=(new_height-1)*a+c-height;
     pad_needed_width=(new_width-1)*b+d-width;
     
    }
    if(strcmp(padding,"VALID")==0)
    {
   
     new_height= (height-c+1+a-1)/a;
     new_width=(width-d+1+b-1)/b;
     pad_needed_height=0;
     pad_needed_width=0;
  
    }  

    float *In,*bias,*Res_In,*W,*output;
    size_t size_1,size_2,size_3,size_4;
    size_1=height*width*batchsize*In_channel* sizeof(float);
    size_2=new_height*new_width*batchsize*In_channel*c*d*sizeof(float);
    size_3=In_channel*c*d*output_channel*sizeof(float);
    size_4=new_height*new_width*batchsize*output_channel*sizeof(float);

    hipMalloc((void**)&In,size_1);  
    hipMalloc((void**)&Res_In,size_2); 
    hipMemcpy(In,A , size_1, hipMemcpyHostToDevice);


    Im2col<< <BLOCK_NUM,THREAD_NUM>> >(In,Res_In,a,b,c,d,height,width,batchsize,In_channel,output_channel,pad_needed_height,pad_needed_width,new_height,new_width);
    hipDeviceSynchronize(); 


    hipMalloc((void**)&W,size_3); 
    hipMalloc((void**)&bias,output_channel*sizeof(float));
    hipMalloc((void**)&output,size_4);
    hipMemcpy(W,B , size_3, hipMemcpyHostToDevice);
    hipMemcpy(bias,C ,output_channel*sizeof(float), hipMemcpyHostToDevice);
    int L_rows=new_height*new_width*batchsize,L_cols=In_channel*c*d,R_cols=output_channel;
    /*
    dim3 dimBlock(blocksize, blocksize);
    OutputMatrix<< <BLOCK_NUM,dimBlock>> >(Res_In,W,bias,output,L_rows,L_cols,R_cols);
    */

    float alpha=1,beta=0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,L_rows,R_cols,L_cols,&alpha,Res_In, L_rows,W,L_cols,&beta,output,L_rows);
    hipblasDestroy(handle);


    //hipDeviceSynchronize(); 

    //�����
    float *Active_output;
    char *Activfun=mxArrayToString(prhs[5]);
    hipMalloc((void**)&Active_output,size_4);

    if(strcmp(Activfun,"Sigmod")==0)
      Sigmod<< <BLOCK_NUM,THREAD_NUM>> >(output,Active_output,new_height,new_width,batchsize,output_channel,bias);

    if(strcmp(Activfun,"Relu")==0)
      Relu<< <BLOCK_NUM,THREAD_NUM>> >(output,Active_output,new_height,new_width,batchsize,output_channel,bias);

    const size_t dim[]={new_height ,new_width,batchsize, output_channel};
    plhs[0] = mxCreateNumericArray(4,dim ,mxSINGLE_CLASS, mxREAL);
    hipMemcpy((float*)mxGetPr(plhs[0]), Active_output, size_4, hipMemcpyDeviceToHost);

    hipFree(In);
    hipFree(Res_In);
    hipFree(W);
    hipFree(bias);
    hipFree(output);
    hipFree(Active_output);

}






